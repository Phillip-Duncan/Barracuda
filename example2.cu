#include "example2.cuh"





int main() 
{
    int threads = 256;
    int blocks = 256;
    dim3 Grid(blocks,1,1);
    dim3 Block(1,threads,1);


    float values[3] = {5,6,10};
    long ops[7] = {16128,0x3CE,0x12FD,0x3CE,0x12FD,0x7E3,0x3CC};
    int stack[10] = {-2,0,0,0,0,0,0,1,1,1};
    float output[6*threads*blocks] =   {0};

    // Allocate some memory for stack expressions
    int* stack_dev = NULL;
    int stacksize = 10;
    long* opstack_dev = NULL;
    long opstacksize = 7;
    float* valuesstack_dev = NULL;
    int valuestacksize = 3;
    float* outputstack_dev = NULL;
    int outputstacksize = 0;

    hipMalloc((void**)&stack_dev,stacksize*sizeof(int));
    hipMemcpy(stack_dev,stack,stacksize*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&opstack_dev,opstacksize*sizeof(long));
    hipMemcpy(opstack_dev,ops,opstacksize*sizeof(long),hipMemcpyHostToDevice);

    hipMalloc((void**)&valuesstack_dev,valuestacksize*sizeof(float));
    hipMemcpy(valuesstack_dev,values,valuestacksize*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc((void**)&outputstack_dev,6*threads*blocks*sizeof(float));
    hipMemset(outputstack_dev,0,6*threads*blocks*sizeof(float));


    // Launch example kernel
    typedef std::chrono::high_resolution_clock Clock;
    auto t1 = Clock::now();

    for (int j=0;j<1;j++) {
        example2_kernel<<<Grid,Block>>>(stack_dev,stacksize,opstack_dev,opstacksize,
        valuesstack_dev,valuestacksize,outputstack_dev,outputstacksize,threads*blocks);
        hipDeviceSynchronize();
    }

    auto t2 = Clock::now();

    hipMemcpy(output,outputstack_dev,6*threads*blocks*sizeof(float),hipMemcpyDeviceToHost);

    std::cout << "outputs: ";
    for (int i=0;i<10;i++) {
         std::cout << output[i] << ", ";
    }
    std::cout << std::endl;


    std::cout << "\n Elapsed time: " << (t2-t1).count()/1e9 << " s" << std::endl;
    // Free memory

    hipFree(stack_dev);
    hipFree(opstack_dev);
    hipFree(valuesstack_dev);
    hipFree(outputstack_dev);

}




