/**
 * @file example2.cu
 * @author Phillip Duncan (phillip.duncan-gelder@pg.canterbury.ac.nz)
 * @brief Function pointers example
 * @version 0.1
 * @date 2021-09-05
 * 
 * @copyright Copyright (c) 2021
 * 
 */

#include "example2.cuh"

int main() 
{
    const int threads = 1;
    const int blocks = 1;
    dim3 Grid(blocks,1,1);
    dim3 Block(1,threads,1);


    double values[10] = {0,0,0,0,0,0,0,5,6,10};
    long long ops[10] = {471736,MUL,LDNX0,MUL,LDNX0,SIN,ADD,0,0,0};
    int stack[10] = {-2,0,0,0,0,0,0,1,1,1};
    double output[6*threads*blocks] =   {0};

    // Allocate some memory for stack expressions
    int* stack_dev = NULL;
    int stacksize = 10;
    long long* opstack_dev = NULL;
    double* valuesstack_dev = NULL;
    double* outputstack_dev = NULL;
    int outputstacksize = 0;

    hipMalloc((void**)&stack_dev,stacksize*sizeof(int));
    hipMemcpy(stack_dev,stack,stacksize*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&opstack_dev,stacksize*sizeof(long long));
    hipMemcpy(opstack_dev,ops,stacksize*sizeof(long long),hipMemcpyHostToDevice);

    hipMalloc((void**)&valuesstack_dev,stacksize*sizeof(double));
    hipMemcpy(valuesstack_dev,values,stacksize*sizeof(double),hipMemcpyHostToDevice);

    hipMalloc((void**)&outputstack_dev,6*threads*blocks*sizeof(double));
    hipMemset(outputstack_dev,0,6*threads*blocks*sizeof(double));


    // Allocate some user-space
    int user_space_size = 64*threads*blocks;

    double* user_space_dev = NULL; 
    hipMalloc((void**)&user_space_dev,user_space_size*sizeof(double));
    hipMemset((void**)&user_space_dev,0,user_space_size*sizeof(double));

    // Launch example kernel
    typedef std::chrono::high_resolution_clock Clock;
    auto t1 = Clock::now();

    for (int j=0;j<1;j++) {
        example2_kernel<float><<<Grid,Block>>>(stack_dev,stacksize,opstack_dev,
        valuesstack_dev,outputstack_dev,outputstacksize,threads*blocks, user_space_dev);
        hipDeviceSynchronize();
    }

    auto t2 = Clock::now();

    hipMemcpy(output,outputstack_dev,6*threads*blocks*sizeof(float),hipMemcpyDeviceToHost);

    std::cout << "outputs: ";
    for (int i=0;i<10;i++) {
         std::cout << output[i] << ", ";
    }
    std::cout << std::endl;


    std::cout << "\n Elapsed time: " << (t2-t1).count()/1e9 << " s" << std::endl;
    // Free memory

    hipFree(stack_dev);
    hipFree(opstack_dev);
    hipFree(valuesstack_dev);
    hipFree(outputstack_dev);

}




