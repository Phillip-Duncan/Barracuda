/**
 * @file example2.cu
 * @author Phillip Duncan (phillip.duncan-gelder@pg.canterbury.ac.nz)
 * @brief Function pointers example
 * @version 0.1
 * @date 2021-09-05
 * 
 * @copyright Copyright (c) 2021
 * 
 */

#include "example2.cuh"

int main() 
{
    const int threads = 1;
    const int blocks = 1;
    dim3 Grid(blocks,1,1);
    dim3 Block(1,threads,1);


    float values[10] = {0,0,0,0,0,0,0,5,6,10};
    long long ops[10] = {703360,0x3CE,0x12FD,0x3CE,0x12FD,0x7E4,0x3CC,0,0,0};
    int stack[10] = {-2,0,0,0,0,0,0,1,1,1};
    double output[6*threads*blocks] =   {0};

    // Allocate some memory for stack expressions
    int* stack_dev = NULL;
    int stacksize = 10;
    long long* opstack_dev = NULL;
    int opstacksize = 10;
    float* valuesstack_dev = NULL;
    int valuestacksize = 10;
    double* outputstack_dev = NULL;
    int outputstacksize = 0;

    hipMalloc((void**)&stack_dev,stacksize*sizeof(int));
    hipMemcpy(stack_dev,stack,stacksize*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&opstack_dev,opstacksize*sizeof(long long));
    hipMemcpy(opstack_dev,ops,opstacksize*sizeof(long long),hipMemcpyHostToDevice);

    hipMalloc((void**)&valuesstack_dev,valuestacksize*sizeof(float));
    hipMemcpy(valuesstack_dev,values,valuestacksize*sizeof(float),hipMemcpyHostToDevice);

    hipMalloc((void**)&outputstack_dev,6*threads*blocks*sizeof(double));
    hipMemset(outputstack_dev,0,6*threads*blocks*sizeof(double));


    // Launch example kernel
    typedef std::chrono::high_resolution_clock Clock;
    auto t1 = Clock::now();

    for (int j=0;j<1;j++) {
        example2_kernel<<<Grid,Block>>>(stack_dev,stacksize,opstack_dev,opstacksize,
        valuesstack_dev,valuestacksize,outputstack_dev,outputstacksize,threads*blocks);
        hipDeviceSynchronize();
    }

    auto t2 = Clock::now();

    hipMemcpy(output,outputstack_dev,6*threads*blocks*sizeof(float),hipMemcpyDeviceToHost);

    std::cout << "outputs: ";
    for (int i=0;i<10;i++) {
         std::cout << output[i] << ", ";
    }
    std::cout << std::endl;


    std::cout << "\n Elapsed time: " << (t2-t1).count()/1e9 << " s" << std::endl;
    // Free memory

    hipFree(stack_dev);
    hipFree(opstack_dev);
    hipFree(valuesstack_dev);
    hipFree(outputstack_dev);

}




